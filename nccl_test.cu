#include <stdlib.h>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"

#define CUDACHECK(cmd) do {                         \
  hipError_t err = cmd;                            \
  if (err != hipSuccess) {                         \
    printf("Failed: Cuda error %s:%d '%s'\n",       \
        __FILE__,__LINE__,hipGetErrorString(err)); \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

#define NCCLCHECK(cmd) do {                         \
  ncclResult_t res = cmd;                           \
  if (res != ncclSuccess) {                         \
    printf("Failed, NCCL error %s:%d '%s'\n",       \
        __FILE__,__LINE__,ncclGetErrorString(res)); \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

int main(int argc, char* argv[])
{
  ncclComm_t comms[2];

  int nDev = 4;
  int size = 32 * 1024 * 1024;
  int devs[4] = { 0, 1, 2, 3};

  // Allocate device buffers
  float *sendbuff[nDev], *recvbuff[nDev];
  hipStream_t s[nDev];

  // Allocate host buffers for debugging
  float *hostSendBuff[nDev], *hostRecvBuff[nDev];
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(devs[i]));
    CUDACHECK(hipMalloc((void**)&sendbuff[i], size * sizeof(float)));
    CUDACHECK(hipMalloc((void**)&recvbuff[i], size * sizeof(float)));
    CUDACHECK(hipStreamCreate(&s[i]));

    hostSendBuff[i] = (float*)malloc(size * sizeof(float));
    hostRecvBuff[i] = (float*)malloc(size * sizeof(float));

    // GPU 0: 1.0f, GPU 1: 2.0f, GPU 2: 3.0f, GPU 3: 4.0f
    for (int j = 0; j < size; ++j) hostSendBuff[i][j] = (float)(i + 1);

    // Copy initialized values to device memory
    CUDACHECK(hipMemcpy(sendbuff[i], hostSendBuff[i], size * sizeof(float), hipMemcpyHostToDevice));
    CUDACHECK(hipMemset(recvbuff[i], 0, size * sizeof(float))); // Ensure recvbuff is cleared
  }

  NCCLCHECK(ncclCommInitAll(comms, nDev, devs));

  // // Verify sendbuff before NCCL operation
  // for (int i = 0; i < nDev; ++i) {
  //   CUDACHECK(cudaMemcpy(hostSendBuff[i], sendbuff[i], size * sizeof(float), cudaMemcpyDeviceToHost));
  //   printf("GPU %d sendBuff first 10 values: ", i);
  //   for (int j = 0; j < 10; j++) printf("%.1f ", hostSendBuff[i][j]);
  //   printf("\n");
  // }

  NCCLCHECK(ncclGroupStart());
  for (int i = 0; i < nDev; ++i) {
    NCCLCHECK(ncclAllReduce((const void*)sendbuff[i], (void*)recvbuff[i], size, ncclFloat, ncclSum,
        comms[i], s[i]));
  }
  NCCLCHECK(ncclGroupEnd());

  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(devs[i]));
    CUDACHECK(hipStreamSynchronize(s[i]));
  }

  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipMemcpy(hostRecvBuff[i], recvbuff[i], size * sizeof(float), hipMemcpyDeviceToHost));

    // Print first 10 elements of recvBuff
    printf("GPU %d recvBuff first 10 values: ", i);
    for (int j = 0; j < 10; j++) {
      printf("%.1f ", hostRecvBuff[i][j]);
    }
    printf("\n");
  }

  // Free device and host buffers
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(devs[i]));
    CUDACHECK(hipFree(sendbuff[i]));
    CUDACHECK(hipFree(recvbuff[i]));
    free(hostSendBuff[i]);
    free(hostRecvBuff[i]);
  }

  for (int i = 0; i < nDev; ++i)
      ncclCommDestroy(comms[i]);

  printf("Success\n");
  return 0;
}
